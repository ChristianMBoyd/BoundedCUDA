#include "hip/hip_runtime.h"
#include "Loss.cuh"

Loss::Loss()
{
	initializeDevice();
}

__global__ void Loss::initializeDevice()
{
	// set device for CUDA use
	hipError_t status = hipSetDevice(0);
	if (status != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Check GPU configuration.");
	}
}

void Loss::Hello()
{
	printf("Hello");
}

__global__ void Loss::addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}