#include "hip/hip_runtime.h"
﻿
#include "Loss.h"

int main()
{
    const int arraySize = 4; // "unsigned int" from cpp POV
    const cuda::std::complex<double> arg[arraySize] = { cuda::std::complex<double>(1,1),
    cuda::std::complex<double>(-1,1) ,cuda::std::complex<double>(1,-1) ,cuda::std::complex<double>(-1,-1) };
    cuda::std::complex<double> root[arraySize] = { 0 };

    Loss g; // initialize GPU and CUDA methods
    g.deviceQuery(); // print out basic GPU facts

    // To do:
    //  1) Implement mathematical functions -- last did Pi0Qn() and Pi0Qnp()
    //  2) Update error-checking - implement function or use try/catch
    //  3) Implement parallelized matrix/vector initializations
    //      i.e., implement in a way that allows for direct cuBLAS calls after
    //  4) Implement cuBLAS methods for subsequent linear algebra

    // current test -- posRoot on complex values, done on device
    hipError_t cudaStatus = g.posRootWithCuda(arg, root, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sqrtWithCuda failed!");
        return 1;
    }

    std::cout << "The result of sqrt({";
    for (int i = 0; i < arraySize; i++)
    {
        std::cout << "(" << arg[i].real() << "," << arg[i].imag() << "), ";
    }
    std::cout << "}):\n";
    std::cout << "{";
    for (int i = 0; i < arraySize; i++)
    {
        std::cout << "(" << root[i].real() << "," << root[i].imag() << "), ";
    }
    std::cout << "})" << std::endl;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // closing preamble to have window hang after printing results
    char input;
    std::cout << "\nEnter any input to close.\n";
    std::cin >> input;

    return 0;
}
